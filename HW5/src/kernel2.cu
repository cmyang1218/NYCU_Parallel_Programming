#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int *deviceData, float lowerX, float lowerY, float stepX, float stepY, int pitch, int count) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re, z_im = c_im;

    int i;
    for (i = 0; i < count; i++) 
    {
    
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    *((int *)((char *)deviceData + pitch * thisY) + thisX) = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    size_t imgSize = resX * resY;
    int *hostData, *deviceData;
    hipHostAlloc((void **)&hostData, imgSize * sizeof(int), hipHostMallocMapped);
    // returns pitch
    size_t pitch = 0;
    hipMallocPitch((void **)&deviceData, &pitch, resX * sizeof(int), resY);
    
    dim3 threadsPerBlock(8, 8);
    dim3 numBlock(resX / threadsPerBlock.x, resY / threadsPerBlock.y);
    
    mandelKernel<<<numBlock, threadsPerBlock>>>(deviceData, lowerX, lowerY, stepX, stepY, pitch, maxIterations);

    hipMemcpy2D(hostData, resX * sizeof(int), deviceData, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, hostData, imgSize * sizeof(int));

    hipFree(deviceData);
    hipHostFree(hostData);
}
